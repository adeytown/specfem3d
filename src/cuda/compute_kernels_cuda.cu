#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <mpi.h>
#include <sys/types.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// ELASTIC SIMULATIONS

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_cudakernel(int* ispec_is_elastic, int* ibool,
					   float* accel,
					   float* b_displ,
					   float* epsilondev_xx,  
					   float* epsilondev_yy,  
					   float* epsilondev_xy,  
					   float* epsilondev_xz,  
					   float* epsilondev_yz,  
					   float* b_epsilondev_xx,
					   float* b_epsilondev_yy,
					   float* b_epsilondev_xy,
					   float* b_epsilondev_xz,
					   float* b_epsilondev_yz,
					   float* rho_kl,					   
					   float deltat,
					   float* mu_kl,
					   float* kappa_kl,
					   float* epsilon_trace_over_3,
					   float* b_epsilon_trace_over_3,
					   int NSPEC_AB,
					   float* d_debug) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;
  if(ispec<NSPEC_AB) { // handles case when there is 1 extra block (due to rectangular grid)
    int ijk = threadIdx.x;
    int ijk_ispec = ijk + 125*ispec;
    int iglob = ibool[ijk_ispec]-1;

    // if(ispec_is_elastic[ispec]) { // leave out until have acoustic coupling
    if(1) {
      
      
      if(ijk_ispec == 9480531) {
      	d_debug[0] = rho_kl[ijk_ispec];
      	d_debug[1] = accel[3*iglob];
      	d_debug[2] = b_displ[3*iglob];
	d_debug[3] = deltat * (accel[3*iglob]*b_displ[3*iglob]+
      				     accel[3*iglob+1]*b_displ[3*iglob+1]+
      				     accel[3*iglob+2]*b_displ[3*iglob+2]);
      }
      
      rho_kl[ijk_ispec] += deltat * (accel[3*iglob]*b_displ[3*iglob]+
      				     accel[3*iglob+1]*b_displ[3*iglob+1]+
      				     accel[3*iglob+2]*b_displ[3*iglob+2]);

      
      
      // if(rho_kl[ijk_ispec] < 1.9983e+18) {
      // atomicAdd(&d_debug[3],1.0);
      // d_debug[4] = ijk_ispec;
	// d_debug[0] = rho_kl[ijk_ispec];
	// d_debug[1] = accel[3*iglob];
	// d_debug[2] = b_displ[3*iglob];
      // }
      
      mu_kl[ijk_ispec] += deltat * (epsilondev_xx[ijk_ispec]*b_epsilondev_xx[ijk_ispec]+ // 1*b1
				    epsilondev_yy[ijk_ispec]*b_epsilondev_yy[ijk_ispec]+ // 2*b2
				    (epsilondev_xx[ijk_ispec]+epsilondev_yy[ijk_ispec])*
				    (b_epsilondev_xx[ijk_ispec]+b_epsilondev_yy[ijk_ispec])+
				    2*(epsilondev_xy[ijk_ispec]*b_epsilondev_xy[ijk_ispec]+
				       epsilondev_xz[ijk_ispec]*b_epsilondev_xz[ijk_ispec]+
				       epsilondev_yz[ijk_ispec]*b_epsilondev_yz[ijk_ispec]));
      
      kappa_kl[ijk_ispec] += deltat*(9*epsilon_trace_over_3[ijk_ispec]*
				     b_epsilon_trace_over_3[ijk_ispec]);
    
    }
  }
}
					   
/* ----------------------------------------------------------------------------------------------- */					   

extern "C" 
void FC_FUNC_(compute_kernels_cuda,
              COMPUTE_KERNELS_CUDA)(long* Mesh_pointer, int* NOISE_TOMOGRAPHY,
                                    int* ELASTIC_SIMULATION, int* SAVE_MOHO_MESH,float* deltat) {

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int blocksize = 125; // NGLLX*NGLLY*NGLLZ
  int num_blocks_x = mp->NSPEC_AB;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = ceil(num_blocks_x/2.0);
    num_blocks_y = num_blocks_y*2;
  }
  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);
  
  float* d_debug;
  float* h_debug;
  h_debug = (float*)calloc(128,sizeof(float));
  hipMalloc((void**)&d_debug,128*sizeof(float));
  hipMemcpy(d_debug,h_debug,128*sizeof(float),hipMemcpyHostToDevice);
  
  
  compute_kernels_cudakernel<<<grid,threads>>>(mp->d_ispec_is_elastic,mp->d_ibool,
					       mp->d_accel, mp->d_b_displ,
					       mp->d_epsilondev_xx,
					       mp->d_epsilondev_yy,
					       mp->d_epsilondev_xy,
					       mp->d_epsilondev_xz,
					       mp->d_epsilondev_yz,
					       mp->d_b_epsilondev_xx,
					       mp->d_b_epsilondev_yy,
					       mp->d_b_epsilondev_xy,
					       mp->d_b_epsilondev_xz,
					       mp->d_b_epsilondev_yz,
					       mp->d_rho_kl,
					       *deltat,
					       mp->d_mu_kl,
					       mp->d_kappa_kl,
					       mp->d_epsilon_trace_over_3,
					       mp->d_b_epsilon_trace_over_3,
					       mp->NSPEC_AB,
					       d_debug);

  hipMemcpy(h_debug,d_debug,128*sizeof(float),hipMemcpyDeviceToHost);
  hipFree(d_debug);
  // for(int i=0;i<5;i++) {
  // printf("d_debug[%d]=%e\n",i,h_debug[i]);
  // }
  free(h_debug);
  
  // float* h_rho = (float*)malloc(sizeof(float)*mp->NSPEC_AB*125);
  // float maxval = 0;
  // hipMemcpy(h_rho,mp->d_rho_kl,sizeof(float)*mp->NSPEC_AB*125,hipMemcpyDeviceToHost);
  // int number_big_values = 0;
  // for(int i=0;i<mp->NSPEC_AB*125;i++) {
  // maxval = MAX(maxval,fabsf(h_rho[i]));
  // if(fabsf(h_rho[i]) > 1e10) {
  // number_big_values++;
  // }
  // }
  
  // printf("maval rho = %e, number>1e10 = %d vs. %d\n",maxval,number_big_values,mp->NSPEC_AB*125);
  
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_cudakernel");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C" 
void FC_FUNC_(transfer_sensitivity_kernels_to_host,
              TRANSFER_SENSITIVITY_KERNELS_TO_HOST)(long* Mesh_pointer, float* h_rho_kl,
                                                    float* h_mu_kl, float* h_kappa_kl,
                                                    float* h_Sigma_kl,int* NSPEC_AB,int* NSPEC_AB_VAL) {

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(h_rho_kl,mp->d_rho_kl,*NSPEC_AB*125*sizeof(float),
				     hipMemcpyDeviceToHost),1);
  print_CUDA_error_if_any(hipMemcpy(h_mu_kl,mp->d_mu_kl,*NSPEC_AB*125*sizeof(float),
				     hipMemcpyDeviceToHost),1);
  print_CUDA_error_if_any(hipMemcpy(h_kappa_kl,mp->d_kappa_kl,*NSPEC_AB*125*sizeof(float),
				     hipMemcpyDeviceToHost),1);
  print_CUDA_error_if_any(hipMemcpy(h_Sigma_kl,mp->d_Sigma_kl,125*(*NSPEC_AB_VAL)*sizeof(float),
				     hipMemcpyHostToDevice),4);
  
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_strength_noise_cuda_kernel(float* displ, 
                                                           int* free_surface_ispec,
                                                           int* free_surface_ijk,
                                                           int* ibool, 
                                                           float* noise_surface_movie, 
                                                           float* normal_x_noise, 
                                                           float* normal_y_noise, 
                                                           float* normal_z_noise, 
                                                           float* Sigma_kl, 
                                                           float deltat,
                                                           int num_free_surface_faces, 
                                                           float* d_debug) {
  int iface = blockIdx.x + blockIdx.y*gridDim.x;
  if(iface<num_free_surface_faces) {

    int ispec = free_surface_ispec[iface]-1;
    int igll = threadIdx.x;        
    int ipoin = igll + 25*iface;
    int i = free_surface_ijk[INDEX3(3,25,0,igll,iface)]-1;
    int j = free_surface_ijk[INDEX3(3,25,0,igll,iface)]-1;
    int k = free_surface_ijk[INDEX3(3,25,0,igll,iface)]-1;
    
    int iglob = ibool[INDEX4(5,5,5,i,j,k,ispec)]-1;
    
    float eta = (noise_surface_movie[INDEX3(3,25,0,igll,iface)]*normal_x_noise[ipoin]+
		 noise_surface_movie[INDEX3(3,25,1,igll,iface)]*normal_y_noise[ipoin]+ 
		 noise_surface_movie[INDEX3(3,25,2,igll,iface)]*normal_z_noise[ipoin]);

    // if(ijk_ispec == 78496) {
    //   d_debug[0] = Sigma_kl[ijk_ispec];
    //   d_debug[1] = eta;
    //   d_debug[2] = normal_x_noise[ipoin];
    //   d_debug[3] = normal_y_noise[ipoin];
    //   d_debug[4] = normal_z_noise[ipoin];
    //   d_debug[5] = displ[3*iglob+2];      
    //   d_debug[6] = deltat*eta*normal_z_noise[ipoin]*displ[2+3*iglob];
    //   d_debug[7] = 0.008*1.000000e-24*normal_z_noise[ipoin]*3.740546e-13;
    // }
    
    Sigma_kl[INDEX4(5,5,5,i,j,k,ispec)] += deltat*eta*(normal_x_noise[ipoin]*displ[3*iglob]+
				       normal_y_noise[ipoin]*displ[1+3*iglob]+
				       normal_z_noise[ipoin]*displ[2+3*iglob]);
  }
    
}

/* ----------------------------------------------------------------------------------------------- */

extern "C" 
void FC_FUNC_(compute_kernels_strength_noise_cuda,
              COMPUTE_KERNELS_STRENGTH_NOISE_CUDA)(long* Mesh_pointer, 
                                                    float* h_noise_surface_movie,
                                                    int* num_free_surface_faces_f,
                                                    float* deltat) {
  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  int num_free_surface_faces = *num_free_surface_faces_f;

  hipMemcpy(mp->d_noise_surface_movie,h_noise_surface_movie,3*25*num_free_surface_faces*sizeof(float),hipMemcpyHostToDevice);


  int num_blocks_x = num_free_surface_faces;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = ceil(num_blocks_x/2.0);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(25,1,1);

  // float* h_debug = (float*)calloc(128,sizeof(float));
  float* d_debug;
  // hipMalloc((void**)&d_debug,128*sizeof(float));
  // hipMemcpy(d_debug,h_debug,128*sizeof(float),hipMemcpyHostToDevice);
  
  compute_kernels_strength_noise_cuda_kernel<<<grid,threads>>>(mp->d_displ,
							       mp->d_free_surface_ispec,
							       mp->d_free_surface_ijk,
							       mp->d_ibool,
							       mp->d_noise_surface_movie,
							       mp->d_normal_x_noise,
							       mp->d_normal_y_noise,
							       mp->d_normal_z_noise,
							       mp->d_Sigma_kl,*deltat,
							       num_free_surface_faces,
							       d_debug);

  // hipMemcpy(h_debug,d_debug,128*sizeof(float),hipMemcpyDeviceToHost);
  // for(int i=0;i<8;i++) {
  //   printf("debug[%d]= %e\n",i,h_debug[i]);
  // }
  
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_strength_noise_cuda_kernel");
#endif
  
}



/* ----------------------------------------------------------------------------------------------- */

// ACOUSTIC SIMULATIONS

/* ----------------------------------------------------------------------------------------------- */


__device__ void compute_gradient_kernel(int ijk,
                                        int ispec,
                                        float* scalar_field,
                                        float* vector_field_element,
                                        float* hprime_xx,
                                        float* hprime_yy,
                                        float* hprime_zz,
                                        float* d_xix, 
                                        float* d_xiy, 
                                        float* d_xiz, 
                                        float* d_etax, 
                                        float* d_etay, 
                                        float* d_etaz, 
                                        float* d_gammax, 
                                        float* d_gammay, 
                                        float* d_gammaz,
                                        float rhol) {
  
  float temp1l,temp2l,temp3l;
  float hp1,hp2,hp3;
  float xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl;
  float rho_invl;
  int l,offset,offset1,offset2,offset3;
  
  //const int NGLLX = 5;
  const int NGLL3_ALIGN = 128;
  
  int K = (ijk/NGLL2);
  int J = ((ijk-K*NGLL2)/NGLLX);
  int I = (ijk-K*NGLL2-J*NGLLX);
  
  // derivative along x
  temp1l = 0.f;
  for( l=0; l<NGLLX;l++){
    hp1 = hprime_xx[l*NGLLX+I];
    offset1 = K*NGLL2+J*NGLLX+l;
    temp1l += scalar_field[offset1]*hp1;
  }
  
  // derivative along y
  temp2l = 0.f;
  for( l=0; l<NGLLX;l++){
    hp2 = hprime_yy[l*NGLLX+J];
    offset2 = K*NGLL2+l*NGLLX+I;
    temp2l += scalar_field[offset2]*hp2;
  }
  
  // derivative along z    
  temp3l = 0.f;
  for( l=0; l<NGLLX;l++){
    hp3 = hprime_zz[l*NGLLX+K];
    offset3 = l*NGLL2+J*NGLLX+I;
    temp3l += scalar_field[offset3]*hp3;
    
  }
  
  offset = ispec*NGLL3_ALIGN + ijk;
  
  xixl = d_xix[offset];
  xiyl = d_xiy[offset];
  xizl = d_xiz[offset];
  etaxl = d_etax[offset];
  etayl = d_etay[offset];
  etazl = d_etaz[offset];
  gammaxl = d_gammax[offset];
  gammayl = d_gammay[offset];
  gammazl = d_gammaz[offset];
  
  rho_invl = 1.0f / rhol;
  
  // derivatives of acoustic scalar potential field on GLL points
  vector_field_element[0] = (temp1l*xixl + temp2l*etaxl + temp3l*gammaxl) * rho_invl;
  vector_field_element[1] = (temp1l*xiyl + temp2l*etayl + temp3l*gammayl) * rho_invl;
  vector_field_element[2] = (temp1l*xizl + temp2l*etazl + temp3l*gammazl) * rho_invl;  
  
}

/* ----------------------------------------------------------------------------------------------- */


__global__ void compute_kernels_acoustic_kernel(int* ispec_is_acoustic, 
                                                int* ibool,
                                                float* rhostore,
                                                float* kappastore,
                                                float* hprime_xx,
                                                float* hprime_yy,
                                                float* hprime_zz,
                                                float* d_xix, 
                                                float* d_xiy, 
                                                float* d_xiz, 
                                                float* d_etax, 
                                                float* d_etay, 
                                                float* d_etaz, 
                                                float* d_gammax, 
                                                float* d_gammay, 
                                                float* d_gammaz,                                                
                                                float* potential_dot_dot_acoustic,
                                                float* b_potential_acoustic,
                                                float* b_potential_dot_dot_acoustic,
                                                float* rho_ac_kl,					   
                                                float* kappa_ac_kl,
                                                float deltat,
                                                int NSPEC_AB) {
  
  int ispec = blockIdx.x + blockIdx.y*gridDim.x;
  int ijk = threadIdx.x;
  
  // local and global indices
  int ijk_ispec = ijk + 125*ispec;
  int ijk_ispec_padded = ijk + 128*ispec;
  int iglob = ibool[ijk_ispec]-1;
  
  float accel_elm[3];
  float b_displ_elm[3];
  float rhol,kappal;
  
  // shared memory between all threads within this block
  __shared__ float scalar_field_displ[125];    
  __shared__ float scalar_field_accel[125];    
  
  if( ispec < NSPEC_AB ){
    if( ispec_is_acoustic[ispec] == 1) { 
      
      // copy field values
      scalar_field_displ[ijk] = b_potential_acoustic[iglob];
      scalar_field_accel[ijk] = potential_dot_dot_acoustic[iglob];
      __syncthreads();
      
      // gets material parameter
      rhol = rhostore[ijk_ispec_padded];
      
      // displacement vector from backward field
      compute_gradient_kernel(ijk,ispec,scalar_field_displ,b_displ_elm,
                              hprime_xx,hprime_yy,hprime_zz,
                              d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                              rhol);
      
      // acceleration vector
      compute_gradient_kernel(ijk,ispec,scalar_field_accel,accel_elm,
                              hprime_xx,hprime_yy,hprime_zz,
                              d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                              rhol);
      
      // density kernel
      rho_ac_kl[ijk_ispec] -= deltat * rhol * (accel_elm[0]*b_displ_elm[0] +
                                               accel_elm[1]*b_displ_elm[1] +
                                               accel_elm[2]*b_displ_elm[2]);
      
      // bulk modulus kernel
      kappal = kappastore[ijk_ispec];
      kappa_ac_kl[ijk_ispec] -= deltat / kappal * potential_dot_dot_acoustic[iglob] 
      * b_potential_dot_dot_acoustic[iglob];    
    }
  }  
}

/* ----------------------------------------------------------------------------------------------- */


extern "C" 
void FC_FUNC_(compute_kernels_acoustic_cuda,
              COMPUTE_KERNELS_ACOUSTIC_CUDA)(
                                             long* Mesh_pointer, 
                                             float* deltat) {
  
TRACE("compute_kernels_acoustic_cuda");
  
  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  
  int blocksize = 125; // NGLLX*NGLLY*NGLLZ
  int num_blocks_x = mp->NSPEC_AB;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = ceil(num_blocks_x/2.0);
    num_blocks_y = num_blocks_y*2;
  }
  
  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);
  
  compute_kernels_acoustic_kernel<<<grid,threads>>>(mp->d_ispec_is_acoustic,
                                                    mp->d_ibool,
                                                    mp->d_rhostore,
                                                    mp->d_kappastore,
                                                    mp->d_hprime_xx,
                                                    mp->d_hprime_yy,
                                                    mp->d_hprime_zz,                                                    
                                                    mp->d_xix, 
                                                    mp->d_xiy, 
                                                    mp->d_xiz,
                                                    mp->d_etax, 
                                                    mp->d_etay, 
                                                    mp->d_etaz,
                                                    mp->d_gammax, 
                                                    mp->d_gammay, 
                                                    mp->d_gammaz,                                                    
                                                    mp->d_potential_dot_dot_acoustic, 
                                                    mp->d_b_potential_acoustic,
                                                    mp->d_b_potential_dot_dot_acoustic,
                                                    mp->d_rho_ac_kl,
                                                    mp->d_kappa_ac_kl,
                                                    *deltat,
                                                    mp->NSPEC_AB);
  
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_acoustic_kernel");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C" 
void FC_FUNC_(transfer_sensitivity_kernels_acoustic_to_host,
              TRANSFER_SENSITIVITY_KERNELS_ACOUSTIC_TO_HOST)(long* Mesh_pointer, 
                                                             float* h_rho_ac_kl,
                                                             float* h_kappa_ac_kl,
                                                             int* NSPEC_AB) {

TRACE("transfer_sensitivity_kernels_acoustic_to_host");  
  
  //get mesh pointer out of fortran integer container  
  Mesh* mp = (Mesh*)(*Mesh_pointer); 
  int size = *NSPEC_AB*125;
  
  // copies kernel values over to CPU host
  print_CUDA_error_if_any(hipMemcpy(h_rho_ac_kl,mp->d_rho_ac_kl,size*sizeof(float),
                                     hipMemcpyDeviceToHost),911);
  print_CUDA_error_if_any(hipMemcpy(h_kappa_ac_kl,mp->d_kappa_ac_kl,size*sizeof(float),
                                     hipMemcpyDeviceToHost),922);  
}
