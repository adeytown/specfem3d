#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 0
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            April 2011
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <mpi.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"
#include "prepare_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// Helper functions

/* ----------------------------------------------------------------------------------------------- */

double get_time()
{
  struct timeval t;
  struct timezone tzp;
  gettimeofday(&t, &tzp);
  return t.tv_sec + t.tv_usec*1e-6;
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(pause_for_debug,PAUSE_FOR_DEBUG)() {
TRACE("pause_for_debug");

  pause_for_debugger(1);
}

/* ----------------------------------------------------------------------------------------------- */

void pause_for_debugger(int pause) {
  if(pause) {
    int myrank;
    MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
    printf("I'm rank %d\n",myrank);
    int i = 0;
    char hostname[256];
    gethostname(hostname, sizeof(hostname));
    printf("PID %d on %s:%d ready for attach\n", getpid(), hostname,myrank);
    FILE *file = fopen("/scratch/eiger/rietmann/attach_gdb.txt","w+");
    fprintf(file,"PID %d on %s:%d ready for attach\n", getpid(), hostname,myrank);
    fclose(file);
    fflush(stdout);
    while (0 == i)
      sleep(5);
  }
}

/* ----------------------------------------------------------------------------------------------- */

void exit_on_cuda_error(char* kernel_name) {
  // sync and check to catch errors from previous async operations
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      fprintf(stderr,"Error after %s: %s\n", kernel_name, hipGetErrorString(err));
      pause_for_debugger(0);
      exit(1);
    }
}

/* ----------------------------------------------------------------------------------------------- */

void exit_on_error(char* info)
{
  printf("\nERROR: %s\n",info);
  fflush(stdout);
#ifdef USE_MPI
  MPI_Abort(MPI_COMM_WORLD,1);
#endif
  exit(EXIT_FAILURE);
  return;
}

/* ----------------------------------------------------------------------------------------------- */

void print_CUDA_error_if_any(hipError_t err, int num)
{
  if (hipSuccess != err)
  {
    printf("\nCUDA error !!!!! <%s> !!!!! \nat CUDA call error code: # %d\n",hipGetErrorString(err),num);
    fflush(stdout);
#ifdef USE_MPI
    MPI_Abort(MPI_COMM_WORLD,1);
#endif
    exit(0);
  }
  return;
}

/* ----------------------------------------------------------------------------------------------- */

void get_free_memory(double* free_db, double* used_db, double* total_db) {

  // gets memory usage in byte
  size_t free_byte ;
  size_t total_byte ;
  hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
  if ( hipSuccess != cuda_status ){
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
    exit(1);
  }

  *free_db = (double)free_byte ;
  *total_db = (double)total_byte ;
  *used_db = *total_db - *free_db ;
  return;
}

/* ----------------------------------------------------------------------------------------------- */

// Saves GPU memory usage to file
void output_free_memory(char* info_str) {
  int myrank;
  MPI_Comm_rank(MPI_COMM_WORLD,&myrank);
  FILE* fp;
  char filename[BUFSIZ];
  double free_db,used_db,total_db;

  get_free_memory(&free_db,&used_db,&total_db);

  sprintf(filename,"../in_out_files/OUTPUT_FILES/gpu_mem_usage_proc_%03d.txt",myrank);
  fp = fopen(filename,"a+");
  fprintf(fp,"%d: @%s GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n", myrank, info_str,
   used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
  fclose(fp);
}

/* ----------------------------------------------------------------------------------------------- */

// Fortran-callable version of above method
extern "C"
void FC_FUNC_(output_free_device_memory,
              OUTPUT_FREE_DEVICE_MEMORY)(int* id) {
TRACE("output_free_device_memory");

  char info[6];
  sprintf(info,"f %d:",*id);
  output_free_memory(info);
}

/* ----------------------------------------------------------------------------------------------- */

void show_free_memory(char* info_str) {

  // show memory usage of GPU
  int myrank;
  MPI_Comm_rank(MPI_COMM_WORLD,&myrank);
  double free_db,used_db,total_db;

  get_free_memory(&free_db,&used_db,&total_db);

  printf("%d: @%s GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n", myrank, info_str,
   used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(show_free_device_memory,
              SHOW_FREE_DEVICE_MEMORY)() {
TRACE("show_free_device_memory");

  show_free_memory("from fortran");
}


extern "C"
void FC_FUNC_(get_free_device_memory,
              get_FREE_DEVICE_MEMORY)(float* free, float* used, float* total ) {
TRACE("get_free_device_memory");

  double free_db,used_db,total_db;

  get_free_memory(&free_db,&used_db,&total_db);

  // converts to MB
  *free = (float) free_db/1024.0/1024.0;
  *used = (float) used_db/1024.0/1024.0;
  *total = (float) total_db/1024.0/1024.0;
  return;
}


/* ----------------------------------------------------------------------------------------------- */
//daniel: helper function
/*
__global__ void check_phase_ispec_kernel(int num_phase_ispec,
                                         int* phase_ispec,
                                         int NSPEC_AB,
                                         int* ier) {

  int i,ispec,iphase,count0,count1;
  *ier = 0;

  for(iphase=0; iphase < 2; iphase++){
    count0 = 0;
    count1 = 0;

    for(i=0; i < num_phase_ispec; i++){
      ispec = phase_ispec[iphase*num_phase_ispec + i] - 1;
      if( ispec < -1 || ispec >= NSPEC_AB ){
        printf("Error in d_phase_ispec_inner_elastic %d %d\n",i,ispec);
        *ier = 1;
        return;
      }
      if( ispec >= 0 ){ count0++;}
      if( ispec < 0 ){ count1++;}
    }

    printf("check_phase_ispec done: phase %d, count = %d %d \n",iphase,count0,count1);

  }
}

void check_phase_ispec(long* Mesh_pointer_f,int type){

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  printf("check phase_ispec for type=%d\n",type);

  dim3 grid(1,1);
  dim3 threads(1,1,1);

  int* h_debug = (int*) calloc(1,sizeof(int));
  int* d_debug;
  hipMalloc((void**)&d_debug,sizeof(int));

  if( type == 1 ){
    check_phase_ispec_kernel<<<grid,threads>>>(mp->num_phase_ispec_elastic,
                                             mp->d_phase_ispec_inner_elastic,
                                             mp->NSPEC_AB,
                                             d_debug);
  }else if( type == 2 ){
    check_phase_ispec_kernel<<<grid,threads>>>(mp->num_phase_ispec_acoustic,
                                               mp->d_phase_ispec_inner_acoustic,
                                               mp->NSPEC_AB,
                                               d_debug);
  }

  hipMemcpy(h_debug,d_debug,1*sizeof(int),hipMemcpyDeviceToHost);
  hipFree(d_debug);
  if( *h_debug != 0 ){printf("error for type=%d\n",type); exit(1);}
  free(h_debug);
  fflush(stdout);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("check_phase_ispec");
#endif

}
*/

/* ----------------------------------------------------------------------------------------------- */
//daniel: helper function
/*
__global__ void check_ispec_is_kernel(int NSPEC_AB,
                                      int* ispec_is,
                                      int* ier) {

  int ispec,count0,count1;

  *ier = 0;
  count0 = 0;
  count1 = 0;
  for(ispec=0; ispec < NSPEC_AB; ispec++){
    if( ispec_is[ispec] < -1 || ispec_is[ispec] > 1 ){
      printf("Error in ispec_is %d %d\n",ispec,ispec_is[ispec]);
      *ier = 1;
      return;
      //exit(1);
    }
    if( ispec_is[ispec] == 0 ){count0++;}
    if( ispec_is[ispec] != 0 ){count1++;}
  }
  printf("check_ispec_is done: count = %d %d\n",count0,count1);
}

void check_ispec_is(long* Mesh_pointer_f,int type){

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  printf("check ispec_is for type=%d\n",type);

  dim3 grid(1,1);
  dim3 threads(1,1,1);

  int* h_debug = (int*) calloc(1,sizeof(int));
  int* d_debug;
  hipMalloc((void**)&d_debug,sizeof(int));

  if( type == 0 ){
    check_ispec_is_kernel<<<grid,threads>>>(mp->NSPEC_AB,
                                            mp->d_ispec_is_inner,
                                            d_debug);
  }else if( type == 1 ){
    check_ispec_is_kernel<<<grid,threads>>>(mp->NSPEC_AB,
                                            mp->d_ispec_is_elastic,
                                            d_debug);
  }else if( type == 2 ){
    check_ispec_is_kernel<<<grid,threads>>>(mp->NSPEC_AB,
                                            mp->d_ispec_is_acoustic,
                                            d_debug);
  }

  hipMemcpy(h_debug,d_debug,1*sizeof(int),hipMemcpyDeviceToHost);
  hipFree(d_debug);
  if( *h_debug != 0 ){printf("error for type=%d\n",type); exit(1);}
  free(h_debug);
  fflush(stdout);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("check_ispec_is");
#endif
}
*/
/* ----------------------------------------------------------------------------------------------- */
//daniel: helper function
/*
__global__ void check_array_ispec_kernel(int num_array_ispec,
                                         int* array_ispec,
                                         int NSPEC_AB,
                                         int* ier) {

  int i,ispec,count0,count1;

  *ier = 0;
  count0 = 0;
  count1 = 0;

  for(i=0; i < num_array_ispec; i++){
    ispec = array_ispec[i] - 1;
    if( ispec < -1 || ispec >= NSPEC_AB ){
      printf("Error in d_array_ispec %d %d\n",i,ispec);
      *ier = 1;
      return;
    }
    if( ispec >= 0 ){ count0++;}
    if( ispec < 0 ){ count1++;}
  }

  printf("check_array_ispec done: count = %d %d \n",count0,count1);
}

void check_array_ispec(long* Mesh_pointer_f,int type){

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  printf("check array_ispec for type=%d\n",type);

  dim3 grid(1,1);
  dim3 threads(1,1,1);

  int* h_debug = (int*) calloc(1,sizeof(int));
  int* d_debug;
  hipMalloc((void**)&d_debug,sizeof(int));

  if( type == 1 ){
    check_array_ispec_kernel<<<grid,threads>>>(mp->d_num_abs_boundary_faces,
                                               mp->d_abs_boundary_ispec,
                                               mp->NSPEC_AB,
                                               d_debug);
  }

  hipMemcpy(h_debug,d_debug,1*sizeof(int),hipMemcpyDeviceToHost);
  hipFree(d_debug);
  if( *h_debug != 0 ){printf("error for type=%d\n",type); exit(1);}
  free(h_debug);
  fflush(stdout);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("check_array_ispec");
#endif

}
*/

/* ----------------------------------------------------------------------------------------------- */

// GPU preparation

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_constants_device,
              PREPARE_CONSTANTS_DEVICE)(long* Mesh_pointer,
                                        int* h_NGLLX,
                                        int* NSPEC_AB, int* NGLOB_AB,
                                        float* h_xix, float* h_xiy, float* h_xiz,
                                        float* h_etax, float* h_etay, float* h_etaz,
                                        float* h_gammax, float* h_gammay, float* h_gammaz,
                                        float* h_kappav, float* h_muv,
                                        int* h_ibool,
                                        int* num_interfaces_ext_mesh,
                                        int* max_nibool_interfaces_ext_mesh,
                                        int* h_nibool_interfaces_ext_mesh,
                                        int* h_ibool_interfaces_ext_mesh,
                                        float* h_hprime_xx,float* h_hprime_yy,float* h_hprime_zz,
                                        float* h_hprimewgll_xx,float* h_hprimewgll_yy,float* h_hprimewgll_zz,
                                        float* h_wgllwgll_xy,float* h_wgllwgll_xz,float* h_wgllwgll_yz,
                                        int* ABSORBING_CONDITIONS,
                                        int* h_abs_boundary_ispec, int* h_abs_boundary_ijk,
                                        float* h_abs_boundary_normal,
                                        float* h_abs_boundary_jacobian2Dw,
                                        int* h_num_abs_boundary_faces,
                                        int* h_ispec_is_inner,
                                        int* NSOURCES,
                                        int* nsources_local_f,
                                        float* h_sourcearrays,
                                        int* h_islice_selected_source,
                                        int* h_ispec_selected_source,
                                        int* h_number_receiver_global,
                                        int* h_ispec_selected_rec,
                                        int* nrec_f,
                                        int* nrec_local_f,
                                        int* SIMULATION_TYPE,
                                        int* USE_MESH_COLORING_GPU_f,
                                        int* nspec_acoustic,int* nspec_elastic,
                                        int* ncuda_devices) {

TRACE("prepare_constants_device");

  int procid;
  int device_count = 0;

  // cuda initialization (needs -lcuda library)
  //hipInit(0);
  hipError_t status = hipInit(0);
  if ( hipSuccess != status ) exit_on_error("CUDA device initialization failed");

  // Gets number of GPU devices
  hipGetDeviceCount(&device_count);
  //printf("Cuda Devices: %d\n", device_count);
  if (device_count == 0) exit_on_error("There is no device supporting CUDA\n");
  *ncuda_devices = device_count;

  // Gets rank number of MPI process
  MPI_Comm_rank(MPI_COMM_WORLD, &procid);

  // Sets the active device
  if(device_count > 1) {
    // generalized for more GPUs per node
    hipSetDevice((procid)%device_count);
    exit_on_cuda_error("hipSetDevice");
  }

  // allocates mesh parameter structure
  Mesh* mp = (Mesh*) malloc( sizeof(Mesh) );
  if (mp == NULL) exit_on_error("error allocating mesh pointer");
  *Mesh_pointer = (long)mp;

  // checks if NGLLX == 5
  if( *h_NGLLX != NGLLX ){
    exit_on_error("NGLLX must be 5 for CUDA devices");
  }

  // sets global parameters
  mp->NSPEC_AB = *NSPEC_AB;
  mp->NGLOB_AB = *NGLOB_AB;

  // sets constant arrays
  setConst_hprime_xx(h_hprime_xx,mp);
  setConst_hprime_yy(h_hprime_yy,mp);
  setConst_hprime_zz(h_hprime_zz,mp);
  setConst_hprimewgll_xx(h_hprimewgll_xx,mp);
  setConst_hprimewgll_yy(h_hprimewgll_yy,mp);
  setConst_hprimewgll_zz(h_hprimewgll_zz,mp);
  setConst_wgllwgll_xy(h_wgllwgll_xy,mp);
  setConst_wgllwgll_xz(h_wgllwgll_xz,mp);
  setConst_wgllwgll_yz(h_wgllwgll_yz,mp);

  /* Assuming NGLLX=5. Padded is then 128 (5^3+3) */
  int size_padded = 128 * (mp->NSPEC_AB);
  int size = 125 * (mp->NSPEC_AB);

  // mesh
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xix, size_padded*sizeof(float)),1001);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiy, size_padded*sizeof(float)),1002);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiz, size_padded*sizeof(float)),1003);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etax, size_padded*sizeof(float)),1004);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etay, size_padded*sizeof(float)),1005);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etaz, size_padded*sizeof(float)),1006);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammax, size_padded*sizeof(float)),1007);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammay, size_padded*sizeof(float)),1008);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammaz, size_padded*sizeof(float)),1009);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_kappav, size_padded*sizeof(float)),1010);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_muv, size_padded*sizeof(float)),1011);

  // transfer constant element data with padding
  for(int i=0;i < mp->NSPEC_AB;i++) {
    print_CUDA_error_if_any(hipMemcpy(mp->d_xix + i*128, &h_xix[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1501);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiy+i*128,   &h_xiy[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1502);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiz+i*128,   &h_xiz[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1503);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etax+i*128,  &h_etax[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1504);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etay+i*128,  &h_etay[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1505);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etaz+i*128,  &h_etaz[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1506);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammax+i*128,&h_gammax[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1507);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammay+i*128,&h_gammay[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1508);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammaz+i*128,&h_gammaz[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1509);
    print_CUDA_error_if_any(hipMemcpy(mp->d_kappav+i*128,&h_kappav[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1510);
    print_CUDA_error_if_any(hipMemcpy(mp->d_muv+i*128,   &h_muv[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),1511);
  }

  // global indexing
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool,size_padded*sizeof(int)),1021);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ibool, h_ibool,
                                     size*sizeof(int),hipMemcpyHostToDevice),1022);


  // prepare interprocess-edge exchange information
  if( *num_interfaces_ext_mesh > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nibool_interfaces_ext_mesh,
                                       (*num_interfaces_ext_mesh)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_nibool_interfaces_ext_mesh,h_nibool_interfaces_ext_mesh,
                                       (*num_interfaces_ext_mesh)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool_interfaces_ext_mesh,
                                       (*num_interfaces_ext_mesh)*(*max_nibool_interfaces_ext_mesh)*sizeof(int)),1203);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibool_interfaces_ext_mesh,h_ibool_interfaces_ext_mesh,
                                       (*num_interfaces_ext_mesh)*(*max_nibool_interfaces_ext_mesh)*sizeof(int),
                                       hipMemcpyHostToDevice),1204);
  }

  // inner elements
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ispec_is_inner,mp->NSPEC_AB*sizeof(int)),1205);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_is_inner, h_ispec_is_inner,
                                     mp->NSPEC_AB*sizeof(int),hipMemcpyHostToDevice),1206);

  // daniel: check
  //check_ispec_is(Mesh_pointer,0);

  // absorbing boundaries
  mp->d_num_abs_boundary_faces = *h_num_abs_boundary_faces;
  if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_abs_boundary_ispec),
                                       (mp->d_num_abs_boundary_faces)*sizeof(int)),1101);
    print_CUDA_error_if_any(hipMemcpy(mp->d_abs_boundary_ispec, h_abs_boundary_ispec,
                                       (mp->d_num_abs_boundary_faces)*sizeof(int),
                                       hipMemcpyHostToDevice),1102);

    // daniel: check
    //check_array_ispec(Mesh_pointer,1);


    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_abs_boundary_ijk),
                                       3*25*(mp->d_num_abs_boundary_faces)*sizeof(int)),1103);
    print_CUDA_error_if_any(hipMemcpy(mp->d_abs_boundary_ijk, h_abs_boundary_ijk,
                                       3*25*(mp->d_num_abs_boundary_faces)*sizeof(int),
                                       hipMemcpyHostToDevice),1104);

    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_abs_boundary_normal),
                                       3*25*(mp->d_num_abs_boundary_faces)*sizeof(float)),1105);
    print_CUDA_error_if_any(hipMemcpy(mp->d_abs_boundary_normal, h_abs_boundary_normal,
                                       3*25*(mp->d_num_abs_boundary_faces)*sizeof(float),
                                       hipMemcpyHostToDevice),1106);

    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_abs_boundary_jacobian2Dw),
                                       25*(mp->d_num_abs_boundary_faces)*sizeof(float)),1107);
    print_CUDA_error_if_any(hipMemcpy(mp->d_abs_boundary_jacobian2Dw, h_abs_boundary_jacobian2Dw,
                                       25*(mp->d_num_abs_boundary_faces)*sizeof(float),
                                       hipMemcpyHostToDevice),1108);
  }

  // sources
  mp->nsources_local = *nsources_local_f;
  if (*SIMULATION_TYPE == 1  || *SIMULATION_TYPE == 3){
    // not needed in case of pure adjoint simulations (SIMULATION_TYPE == 2)
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_sourcearrays,
                                       sizeof(float)* *NSOURCES*3*125),1301);
    print_CUDA_error_if_any(hipMemcpy(mp->d_sourcearrays, h_sourcearrays,
                                       sizeof(float)* *NSOURCES*3*125,hipMemcpyHostToDevice),1302);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_stf_pre_compute,
                                       *NSOURCES*sizeof(double)),1303);
  }

  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_islice_selected_source,
                                     sizeof(int) * *NSOURCES),1401);
  print_CUDA_error_if_any(hipMemcpy(mp->d_islice_selected_source, h_islice_selected_source,
                                     sizeof(int)* *NSOURCES,hipMemcpyHostToDevice),1402);

  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_ispec_selected_source,
                                     sizeof(int)* *NSOURCES),1403);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_selected_source, h_ispec_selected_source,
                                     sizeof(int)* *NSOURCES,hipMemcpyHostToDevice),1404);


  // receiver stations
  int nrec = *nrec_f; // total number of receivers
  mp->nrec_local = *nrec_local_f; // number of receiver located in this partition
  //int nrec_local = *nrec_local_f;
  // note that:
  // size(number_receiver_global) = nrec_local
  // size(ispec_selected_rec) = nrec
  if( mp->nrec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_number_receiver_global),mp->nrec_local*sizeof(int)),1);
    print_CUDA_error_if_any(hipMemcpy(mp->d_number_receiver_global,h_number_receiver_global,
                                     mp->nrec_local*sizeof(int),hipMemcpyHostToDevice),1512);
  }
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ispec_selected_rec),nrec*sizeof(int)),1513);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_selected_rec,h_ispec_selected_rec,
                                     nrec*sizeof(int),hipMemcpyHostToDevice),1514);

#ifdef USE_MESH_COLORING_GPU
  mp->use_mesh_coloring_gpu = 1;
  if( ! *USE_MESH_COLORING_GPU_f ) exit_on_error("error with USE_MESH_COLORING_GPU constant; please re-compile\n");
#else
  // mesh coloring
  // note: this here passes the coloring as an option to the kernel routines
  //          the performance seems to be the same if one uses the pre-processing directives above or not
  mp->use_mesh_coloring_gpu = *USE_MESH_COLORING_GPU_f;
#endif

  // number of elements per domain
  mp->nspec_acoustic = *nspec_acoustic;
  mp->nspec_elastic = *nspec_elastic;

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_constants_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// purely adjoint & kernel simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_sim2_or_3_const_device,
              PREPARE_SIM2_OR_3_CONST_DEVICE)(
                                              long* Mesh_pointer_f,
                                              int* islice_selected_rec,
                                              int* islice_selected_rec_size,
                                              int* nadj_rec_local,
                                              int* nrec,
                                              int* myrank) {

TRACE("prepare_sim2_or_3_const_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // allocates arrays for receivers
  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_islice_selected_rec,
                                     *islice_selected_rec_size*sizeof(int)),7001);
  // copies arrays to GPU device
  print_CUDA_error_if_any(hipMemcpy(mp->d_islice_selected_rec,islice_selected_rec,
                                     *islice_selected_rec_size*sizeof(int),hipMemcpyHostToDevice),7002);

  // adjoint source arrays
  mp->nadj_rec_local = *nadj_rec_local;
  if( mp->nadj_rec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_adj_sourcearrays,
                                       (mp->nadj_rec_local)*3*125*sizeof(float)),7003);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_pre_computed_irec,
                                       (mp->nadj_rec_local)*sizeof(int)),7004);

    // prepares local irec array:
    // the irec_local variable needs to be precomputed (as
    // h_pre_comp..), because normally it is in the loop updating accel,
    // and due to how it's incremented, it cannot be parallelized
    int* h_pre_computed_irec = (int*) malloc( (mp->nadj_rec_local)*sizeof(int) );
    if( h_pre_computed_irec == NULL ) exit_on_error("prepare_sim2_or_3_const_device: h_pre_computed_irec not allocated\n");

    int irec_local = 0;
    for(int irec = 0; irec < *nrec; irec++) {
      if(*myrank == islice_selected_rec[irec]) {
        irec_local++;
        h_pre_computed_irec[irec_local-1] = irec;
      }
    }
    if( irec_local != mp->nadj_rec_local ) exit_on_error("prepare_sim2_or_3_const_device: irec_local not equal\n");
    // copies values onto GPU
    print_CUDA_error_if_any(hipMemcpy(mp->d_pre_computed_irec,h_pre_computed_irec,
                                       (mp->nadj_rec_local)*sizeof(int),hipMemcpyHostToDevice),7010);
    free(h_pre_computed_irec);

    // temporary array to prepare extracted source array values
    mp->h_adj_sourcearrays_slice = (float*) malloc( (mp->nadj_rec_local)*3*125*sizeof(float) );
    if( mp->h_adj_sourcearrays_slice == NULL ) exit_on_error("h_adj_sourcearrays_slice not allocated\n");

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_sim2_or_3_const_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// for ACOUSTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_acoustic_device,
              PREPARE_FIELDS_ACOUSTIC_DEVICE)(long* Mesh_pointer_f,
                                              float* rmass_acoustic,
                                              float* rhostore,
                                              float* kappastore,
                                              int* num_phase_ispec_acoustic,
                                              int* phase_ispec_inner_acoustic,
                                              int* ispec_is_acoustic,
                                              int* NOISE_TOMOGRAPHY,
                                              int* num_free_surface_faces,
                                              int* free_surface_ispec,
                                              int* free_surface_ijk,
                                              int* ABSORBING_CONDITIONS,
                                              int* b_reclen_potential,
                                              float* b_absorb_potential,
                                              int* ELASTIC_SIMULATION,
                                              int* num_coupling_ac_el_faces,
                                              int* coupling_ac_el_ispec,
                                              int* coupling_ac_el_ijk,
                                              float* coupling_ac_el_normal,
                                              float* coupling_ac_el_jacobian2Dw,
                                              int* num_colors_outer_acoustic,
                                              int* num_colors_inner_acoustic,
                                              int* num_elem_colors_acoustic) {

  TRACE("prepare_fields_acoustic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);
  /* Assuming NGLLX==5. Padded is then 128 (5^3+3) */
  int size_padded = 128 * mp->NSPEC_AB;
  int size_nonpadded = 125 * mp->NSPEC_AB;
  int size = mp->NGLOB_AB;

  // allocates arrays on device (GPU)
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_acoustic),sizeof(float)*size),9001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_dot_acoustic),sizeof(float)*size),9002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_dot_dot_acoustic),sizeof(float)*size),9003);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_potential_dot_dot_buffer),sizeof(float)*size),9004);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass_acoustic),sizeof(float)*size),9005);
  // padded array
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rhostore),size_padded*sizeof(float)),9006);
  // non-padded array
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappastore),size_nonpadded*sizeof(float)),9007);

  // transfer element data
  print_CUDA_error_if_any(hipMemcpy(mp->d_rmass_acoustic,rmass_acoustic,
                                     sizeof(float)*size,hipMemcpyHostToDevice),9100);
  print_CUDA_error_if_any(hipMemcpy(mp->d_kappastore,kappastore,
                                     size_nonpadded*sizeof(float),hipMemcpyHostToDevice),9105);
  // transfer constant element data with padding
  for(int i=0; i < mp->NSPEC_AB; i++) {
    print_CUDA_error_if_any(hipMemcpy(mp->d_rhostore+i*128, &rhostore[i*125],
                                       125*sizeof(float),hipMemcpyHostToDevice),9106);
  }

  // phase elements
  mp->num_phase_ispec_acoustic = *num_phase_ispec_acoustic;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_phase_ispec_inner_acoustic),
                                      mp->num_phase_ispec_acoustic*2*sizeof(int)),9008);
  print_CUDA_error_if_any(hipMemcpy(mp->d_phase_ispec_inner_acoustic,phase_ispec_inner_acoustic,
                                     mp->num_phase_ispec_acoustic*2*sizeof(int),hipMemcpyHostToDevice),9101);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ispec_is_acoustic),
                                     mp->NSPEC_AB*sizeof(int)),9009);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_is_acoustic,ispec_is_acoustic,
                                     mp->NSPEC_AB*sizeof(int),hipMemcpyHostToDevice),9102);

  // free surface
  if( *NOISE_TOMOGRAPHY == 0 ){
    // allocate surface arrays
    mp->num_free_surface_faces = *num_free_surface_faces;
    if( mp->num_free_surface_faces > 0 ){
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_ispec),
                                       mp->num_free_surface_faces*sizeof(int)),9201);
      print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ispec,free_surface_ispec,
                                       mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),9203);

      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_ijk),
                                       3*25*mp->num_free_surface_faces*sizeof(int)),9202);
      print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ijk,free_surface_ijk,
                                       3*25*mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),9204);
    }
  }

  // absorbing boundaries
  if( *ABSORBING_CONDITIONS ){
    mp->d_b_reclen_potential = *b_reclen_potential;
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_absorb_potential),mp->d_b_reclen_potential),9301);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_potential,b_absorb_potential,
                                       mp->d_b_reclen_potential,hipMemcpyHostToDevice),9302);
  }


  // for seismograms
  if( mp->nrec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_station_seismo_potential),
                                       mp->nrec_local*125*sizeof(float)),9107);
    mp->h_station_seismo_potential = (float*) malloc( mp->nrec_local*125*sizeof(float) );
    if( mp->h_station_seismo_potential == NULL) exit_on_error("error allocating h_station_seismo_potential");
  }


  // coupling with elastic parts
  if( *ELASTIC_SIMULATION && *num_coupling_ac_el_faces > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_coupling_ac_el_ispec),
                                       (*num_coupling_ac_el_faces)*sizeof(int)),9601);
    print_CUDA_error_if_any(hipMemcpy(mp->d_coupling_ac_el_ispec,coupling_ac_el_ispec,
                                       (*num_coupling_ac_el_faces)*sizeof(int),hipMemcpyHostToDevice),9602);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_coupling_ac_el_ijk),
                                       3*25*(*num_coupling_ac_el_faces)*sizeof(int)),9603);
    print_CUDA_error_if_any(hipMemcpy(mp->d_coupling_ac_el_ijk,coupling_ac_el_ijk,
                                       3*25*(*num_coupling_ac_el_faces)*sizeof(int),hipMemcpyHostToDevice),9604);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_coupling_ac_el_normal),
                                        3*25*(*num_coupling_ac_el_faces)*sizeof(float)),9605);
    print_CUDA_error_if_any(hipMemcpy(mp->d_coupling_ac_el_normal,coupling_ac_el_normal,
                                        3*25*(*num_coupling_ac_el_faces)*sizeof(float),hipMemcpyHostToDevice),9606);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_coupling_ac_el_jacobian2Dw),
                                        25*(*num_coupling_ac_el_faces)*sizeof(float)),9607);
    print_CUDA_error_if_any(hipMemcpy(mp->d_coupling_ac_el_jacobian2Dw,coupling_ac_el_jacobian2Dw,
                                        25*(*num_coupling_ac_el_faces)*sizeof(float),hipMemcpyHostToDevice),9608);

  }

  // mesh coloring
  if( mp->use_mesh_coloring_gpu ){
    mp->num_colors_outer_acoustic = *num_colors_outer_acoustic;
    mp->num_colors_inner_acoustic = *num_colors_inner_acoustic;
    mp->h_num_elem_colors_acoustic = (int*) num_elem_colors_acoustic;
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_acoustic_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_acoustic_adj_dev,
              PREPARE_FIELDS_ACOUSTIC_ADJ_DEV)(long* Mesh_pointer_f,
                                              int* SIMULATION_TYPE,
                                              int* APPROXIMATE_HESS_KL) {

  TRACE("prepare_fields_acoustic_adj_dev");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  int size = mp->NGLOB_AB;

  // kernel simulations
  if( *SIMULATION_TYPE != 3 ) return;

  // allocates backward/reconstructed arrays on device (GPU)
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_acoustic),sizeof(float)*size),9014);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_dot_acoustic),sizeof(float)*size),9015);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_dot_dot_acoustic),sizeof(float)*size),9016);

  // allocates kernels
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_ac_kl),125*mp->NSPEC_AB*sizeof(float)),9017);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappa_ac_kl),125*mp->NSPEC_AB*sizeof(float)),9018);

  // initializes kernel values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_rho_ac_kl,0,
                                     125*mp->NSPEC_AB*sizeof(float)),9019);
  print_CUDA_error_if_any(hipMemset(mp->d_kappa_ac_kl,0,
                                     125*mp->NSPEC_AB*sizeof(float)),9020);

  // preconditioner
  if( *APPROXIMATE_HESS_KL ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_ac_kl),125*mp->NSPEC_AB*sizeof(float)),9030);
    // initializes with zeros
    print_CUDA_error_if_any(hipMemset(mp->d_hess_ac_kl,0,
                                       125*mp->NSPEC_AB*sizeof(float)),9031);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_acoustic_adj_dev");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// for ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_elastic_device,
              PREPARE_FIELDS_ELASTIC_DEVICE)(long* Mesh_pointer_f,
                                             int* size,
                                             float* rmass,
                                             float* rho_vp,
                                             float* rho_vs,
                                             int* num_phase_ispec_elastic,
                                             int* phase_ispec_inner_elastic,
                                             int* ispec_is_elastic,
                                             int* ABSORBING_CONDITIONS,
                                             float* h_b_absorb_field,
                                             int* h_b_reclen_field,
                                             int* SIMULATION_TYPE,int* SAVE_FORWARD,
                                             int* COMPUTE_AND_STORE_STRAIN,
                                             float* epsilondev_xx,float* epsilondev_yy,float* epsilondev_xy,
                                             float* epsilondev_xz,float* epsilondev_yz,
                                             int* ATTENUATION,
                                             int* R_size,
                                             float* R_xx,float* R_yy,float* R_xy,float* R_xz,float* R_yz,
                                             float* one_minus_sum_beta,float* factor_common,
                                             float* alphaval,float* betaval,float* gammaval,
                                             int* OCEANS,
                                             float* rmass_ocean_load,
                                             int* NOISE_TOMOGRAPHY,
                                             float* free_surface_normal,
                                             int* free_surface_ispec,
                                             int* free_surface_ijk,
                                             int* num_free_surface_faces,
                                             int* ACOUSTIC_SIMULATION,
                                             int* num_colors_outer_elastic,
                                             int* num_colors_inner_elastic,
                                             int* num_elem_colors_elastic){

TRACE("prepare_fields_elastic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);
  /* Assuming NGLLX==5. Padded is then 128 (5^3+3) */
  //int size_padded = 128 * mp->NSPEC_AB;
  int size_nonpadded = 125 * mp->NSPEC_AB;

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_displ),sizeof(float)*(*size)),8001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_veloc),sizeof(float)*(*size)),8002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_accel),sizeof(float)*(*size)),8003);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_accel_buffer),sizeof(float)*(*size)),8004);

  // mass matrix
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass),sizeof(float)*mp->NGLOB_AB),8005);
  // transfer element data
  print_CUDA_error_if_any(hipMemcpy(mp->d_rmass,rmass,
                                     sizeof(float)*mp->NGLOB_AB,hipMemcpyHostToDevice),8010);


  // element indices
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ispec_is_elastic),mp->NSPEC_AB*sizeof(int)),8009);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_is_elastic,ispec_is_elastic,
                                     mp->NSPEC_AB*sizeof(int),hipMemcpyHostToDevice),8012);

  // daniel: check
  //check_ispec_is(Mesh_pointer_f,1);

  // phase elements
  mp->num_phase_ispec_elastic = *num_phase_ispec_elastic;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_phase_ispec_inner_elastic),
                                     mp->num_phase_ispec_elastic*2*sizeof(int)),8008);
  print_CUDA_error_if_any(hipMemcpy(mp->d_phase_ispec_inner_elastic,phase_ispec_inner_elastic,
                                     mp->num_phase_ispec_elastic*2*sizeof(int),hipMemcpyHostToDevice),8011);

  //daniel: check
  //check_phase_ispec(Mesh_pointer_f,1);

  // for seismograms
  if( mp->nrec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_station_seismo_field),
                                     3*125*(mp->nrec_local)*sizeof(float)),8015);
    mp->h_station_seismo_field = (float*) malloc( 3*125*(mp->nrec_local)*sizeof(float) );
    if( mp->h_station_seismo_field == NULL) exit_on_error("h_station_seismo_field not allocated \n");
  }

  // absorbing conditions
  if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0){
    // non-padded arrays
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_vp),size_nonpadded*sizeof(float)),8006);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_vs),size_nonpadded*sizeof(float)),8007);

    // rho_vp, rho_vs non-padded; they are needed for stacey boundary condition
    print_CUDA_error_if_any(hipMemcpy(mp->d_rho_vp, rho_vp,
                                       size_nonpadded*sizeof(float),hipMemcpyHostToDevice),8013);
    print_CUDA_error_if_any(hipMemcpy(mp->d_rho_vs, rho_vs,
                                       size_nonpadded*sizeof(float),hipMemcpyHostToDevice),8014);

    // absorb_field array used for file i/o
    if(*SIMULATION_TYPE == 3 || ( *SIMULATION_TYPE == 1 && *SAVE_FORWARD )){
      mp->d_b_reclen_field = *h_b_reclen_field;
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_absorb_field),
                                       mp->d_b_reclen_field),8016);
      print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_field, h_b_absorb_field,
                                       mp->d_b_reclen_field,hipMemcpyHostToDevice),8017);
    }
  }

  // strains used for attenuation and kernel simulations
  if( *COMPUTE_AND_STORE_STRAIN ){
    // strains
    int epsilondev_size = 125*mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_xx,
                                       epsilondev_size*sizeof(float)),8301);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xx,epsilondev_xx,epsilondev_size*sizeof(float),
                                       hipMemcpyHostToDevice),8302);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_yy,
                                       epsilondev_size*sizeof(float)),8302);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_yy,epsilondev_yy,epsilondev_size*sizeof(float),
                                       hipMemcpyHostToDevice),8303);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_xy,
                                       epsilondev_size*sizeof(float)),8304);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xy,epsilondev_xy,epsilondev_size*sizeof(float),
                                       hipMemcpyHostToDevice),8305);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_xz,
                                       epsilondev_size*sizeof(float)),8306);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xz,epsilondev_xz,epsilondev_size*sizeof(float),
                                       hipMemcpyHostToDevice),8307);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_yz,
                                       epsilondev_size*sizeof(float)),8308);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_yz,epsilondev_yz,epsilondev_size*sizeof(float),
                                       hipMemcpyHostToDevice),8309);

  }

  // attenuation memory variables
  if( *ATTENUATION ){
    // memory arrays
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_xx),
                                       (*R_size)*sizeof(float)),8401);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xx,R_xx,(*R_size)*sizeof(float),
                                       hipMemcpyHostToDevice),8402);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_yy),
                                       (*R_size)*sizeof(float)),8403);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_yy,R_yy,(*R_size)*sizeof(float),
                                       hipMemcpyHostToDevice),8404);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_xy),
                                       (*R_size)*sizeof(float)),8405);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xy,R_xy,(*R_size)*sizeof(float),
                                       hipMemcpyHostToDevice),8406);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_xz),
                                       (*R_size)*sizeof(float)),8407);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xz,R_xz,(*R_size)*sizeof(float),
                                       hipMemcpyHostToDevice),8408);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_yz),
                                       (*R_size)*sizeof(float)),8409);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_yz,R_yz,(*R_size)*sizeof(float),
                                       hipMemcpyHostToDevice),8410);

    // attenuation factors
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_one_minus_sum_beta),
                                       125*mp->NSPEC_AB*sizeof(float)),8430);
    print_CUDA_error_if_any(hipMemcpy(mp->d_one_minus_sum_beta ,one_minus_sum_beta,
                                       125*mp->NSPEC_AB*sizeof(float),hipMemcpyHostToDevice),8431);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_factor_common),
                                       N_SLS*125*mp->NSPEC_AB*sizeof(float)),8432);
    print_CUDA_error_if_any(hipMemcpy(mp->d_factor_common ,factor_common,
                                       N_SLS*125*mp->NSPEC_AB*sizeof(float),hipMemcpyHostToDevice),8433);

    // alpha,beta,gamma factors
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_alphaval),
                                       N_SLS*sizeof(float)),8434);
    print_CUDA_error_if_any(hipMemcpy(mp->d_alphaval ,alphaval,
                                       N_SLS*sizeof(float),hipMemcpyHostToDevice),8435);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_betaval),
                                       N_SLS*sizeof(float)),8436);
    print_CUDA_error_if_any(hipMemcpy(mp->d_betaval ,betaval,
                                       N_SLS*sizeof(float),hipMemcpyHostToDevice),8437);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_gammaval),
                                       N_SLS*sizeof(float)),8438);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammaval ,gammaval,
                                       N_SLS*sizeof(float),hipMemcpyHostToDevice),8439);

  }


  if( *OCEANS ){
    // oceans needs a free surface
    mp->num_free_surface_faces = *num_free_surface_faces;
    if( mp->num_free_surface_faces > 0 ){
      // mass matrix
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass_ocean_load),
                                         sizeof(float)*mp->NGLOB_AB),8501);
      print_CUDA_error_if_any(hipMemcpy(mp->d_rmass_ocean_load,rmass_ocean_load,
                                         sizeof(float)*mp->NGLOB_AB,hipMemcpyHostToDevice),8502);
      // surface normal
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_normal),
                                         3*25*(mp->num_free_surface_faces)*sizeof(float)),8503);
      print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_normal,free_surface_normal,
                                         3*25*(mp->num_free_surface_faces)*sizeof(float),hipMemcpyHostToDevice),8504);

      // temporary global array: used to synchronize updates on global accel array
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_updated_dof_ocean_load),
                                         sizeof(int)*mp->NGLOB_AB),8505);

      if( *NOISE_TOMOGRAPHY == 0 && *ACOUSTIC_SIMULATION == 0 ){
        print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_ispec),
                                          mp->num_free_surface_faces*sizeof(int)),9201);
        print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ispec,free_surface_ispec,
                                          mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),9203);
        print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_ijk),
                                          3*25*mp->num_free_surface_faces*sizeof(int)),9202);
        print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ijk,free_surface_ijk,
                                          3*25*mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),9204);
      }
    }
  }

  // mesh coloring
  if( mp->use_mesh_coloring_gpu ){
    mp->num_colors_outer_elastic = *num_colors_outer_elastic;
    mp->num_colors_inner_elastic = *num_colors_inner_elastic;
    mp->h_num_elem_colors_elastic = (int*) num_elem_colors_elastic;
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_elastic_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_elastic_adj_dev,
              PREPARE_FIELDS_ELASTIC_ADJ_DEV)(long* Mesh_pointer_f,
                                             int* size,
                                             int* SIMULATION_TYPE,
                                             int* COMPUTE_AND_STORE_STRAIN,
                                             float* epsilon_trace_over_3,
                                             float* b_epsilondev_xx,float* b_epsilondev_yy,float* b_epsilondev_xy,
                                             float* b_epsilondev_xz,float* b_epsilondev_yz,
                                             float* b_epsilon_trace_over_3,
                                             int* ATTENUATION,
                                             int* R_size,
                                             float* b_R_xx,float* b_R_yy,float* b_R_xy,float* b_R_xz,float* b_R_yz,
                                             float* b_alphaval,float* b_betaval,float* b_gammaval,
                                             int* APPROXIMATE_HESS_KL){

  TRACE("prepare_fields_elastic_adj_dev");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // checks if kernel simulation
  if( *SIMULATION_TYPE != 3 ) return;

  // kernel simulations
  // allocates backward/reconstructed arrays on device (GPU)
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_displ),sizeof(float)*(*size)),8201);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_veloc),sizeof(float)*(*size)),8202);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_accel),sizeof(float)*(*size)),8203);

  // allocates kernels
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_kl),125*mp->NSPEC_AB*sizeof(float)),8204);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_mu_kl),125*mp->NSPEC_AB*sizeof(float)),8205);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappa_kl),125*mp->NSPEC_AB*sizeof(float)),8206);

  // initializes kernel values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_rho_kl,0,
                                     125*mp->NSPEC_AB*sizeof(float)),8207);
  print_CUDA_error_if_any(hipMemset(mp->d_mu_kl,0,
                                     125*mp->NSPEC_AB*sizeof(float)),8208);
  print_CUDA_error_if_any(hipMemset(mp->d_kappa_kl,0,
                                     125*mp->NSPEC_AB*sizeof(float)),8209);

  // strains used for attenuation and kernel simulations
  if( *COMPUTE_AND_STORE_STRAIN ){
    // strains
    int epsilondev_size = 125*mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing

    // solid pressure
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_epsilon_trace_over_3),
                                       125*mp->NSPEC_AB*sizeof(float)),8310);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilon_trace_over_3,epsilon_trace_over_3,
                                       125*mp->NSPEC_AB*sizeof(float),hipMemcpyHostToDevice),8311);
    // backward solid pressure
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilon_trace_over_3),
                                       125*mp->NSPEC_AB*sizeof(float)),8312);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilon_trace_over_3 ,b_epsilon_trace_over_3,
                                       125*mp->NSPEC_AB*sizeof(float),hipMemcpyHostToDevice),8313);
    // prepares backward strains
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilondev_xx),
                                       epsilondev_size*sizeof(float)),8321);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilondev_yy),
                                       epsilondev_size*sizeof(float)),8322);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilondev_xy),
                                       epsilondev_size*sizeof(float)),8323);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilondev_xz),
                                       epsilondev_size*sizeof(float)),8324);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilondev_yz),
                                       epsilondev_size*sizeof(float)),8325);

    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xx,b_epsilondev_xx,
                                       epsilondev_size*sizeof(float),hipMemcpyHostToDevice),8326);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yy,b_epsilondev_yy,
                                       epsilondev_size*sizeof(float),hipMemcpyHostToDevice),8327);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xy,b_epsilondev_xy,
                                       epsilondev_size*sizeof(float),hipMemcpyHostToDevice),8328);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xz,b_epsilondev_xz,
                                       epsilondev_size*sizeof(float),hipMemcpyHostToDevice),8329);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yz,b_epsilondev_yz,
                                       epsilondev_size*sizeof(float),hipMemcpyHostToDevice),8330);
  }

  // attenuation memory variables
  if( *ATTENUATION ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_R_xx),
                                       (*R_size)*sizeof(float)),8421);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xx,b_R_xx,(*R_size)*sizeof(float),
                                       hipMemcpyHostToDevice),8422);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_R_yy),
                                       (*R_size)*sizeof(float)),8423);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_yy,b_R_yy,(*R_size)*sizeof(float),
                                       hipMemcpyHostToDevice),8424);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_R_xy),
                                       (*R_size)*sizeof(float)),8425);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xy,b_R_xy,(*R_size)*sizeof(float),
                                       hipMemcpyHostToDevice),8426);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_R_xz),
                                       (*R_size)*sizeof(float)),8427);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xz,b_R_xz,(*R_size)*sizeof(float),
                                       hipMemcpyHostToDevice),8428);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_R_yz),
                                       (*R_size)*sizeof(float)),8429);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_yz,b_R_yz,(*R_size)*sizeof(float),
                                       hipMemcpyHostToDevice),8420);

    // alpha,beta,gamma factors for backward fields
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_alphaval),
                                       N_SLS*sizeof(float)),8434);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_alphaval ,b_alphaval,
                                       N_SLS*sizeof(float),hipMemcpyHostToDevice),8435);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_betaval),
                                       N_SLS*sizeof(float)),8436);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_betaval ,b_betaval,
                                       N_SLS*sizeof(float),hipMemcpyHostToDevice),8437);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_gammaval),
                                       N_SLS*sizeof(float)),8438);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_gammaval ,b_gammaval,
                                       N_SLS*sizeof(float),hipMemcpyHostToDevice),8439);
  }

  if( *APPROXIMATE_HESS_KL ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_el_kl),125*mp->NSPEC_AB*sizeof(float)),8450);
    // initializes with zeros
    print_CUDA_error_if_any(hipMemset(mp->d_hess_el_kl,0,
                                       125*mp->NSPEC_AB*sizeof(float)),8451);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_elastic_adj_dev");
#endif
}



/* ----------------------------------------------------------------------------------------------- */

// for NOISE simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_noise_device,
              PREPARE_FIELDS_NOISE_DEVICE)(long* Mesh_pointer_f,
                                           int* NSPEC_AB, int* NGLOB_AB,
                                           int* free_surface_ispec,
                                           int* free_surface_ijk,
                                           int* num_free_surface_faces,
                                           int* SIMULATION_TYPE,
                                           int* NOISE_TOMOGRAPHY,
                                           int* NSTEP,
                                           float* noise_sourcearray,
                                           float* normal_x_noise,
                                           float* normal_y_noise,
                                           float* normal_z_noise,
                                           float* mask_noise,
                                           float* free_surface_jacobian2Dw) {

  TRACE("prepare_fields_noise_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // free surface
  mp->num_free_surface_faces = *num_free_surface_faces;

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_free_surface_ispec,
                                     mp->num_free_surface_faces*sizeof(int)),4001);
  print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ispec, free_surface_ispec,
                                     mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),4002);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_free_surface_ijk,
                                     3*25*mp->num_free_surface_faces*sizeof(int)),4003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ijk,free_surface_ijk,
                                     3*25*mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),4004);

  // alloc storage for the surface buffer to be copied
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_noise_surface_movie,
                                     3*25*mp->num_free_surface_faces*sizeof(float)),4005);

  // prepares noise source array
  if( *NOISE_TOMOGRAPHY == 1 ){
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_noise_sourcearray,
                                       3*125*(*NSTEP)*sizeof(float)),4101);
    print_CUDA_error_if_any(hipMemcpy(mp->d_noise_sourcearray, noise_sourcearray,
                                       3*125*(*NSTEP)*sizeof(float),hipMemcpyHostToDevice),4102);
  }

  // prepares noise directions
  if( *NOISE_TOMOGRAPHY > 1 ){
    int nface_size = 25*(*num_free_surface_faces);
    // allocates memory on GPU
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_normal_x_noise,
                                       nface_size*sizeof(float)),4301);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_normal_y_noise,
                                       nface_size*sizeof(float)),4302);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_normal_z_noise,
                                       nface_size*sizeof(float)),4303);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_mask_noise,
                                       nface_size*sizeof(float)),4304);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_free_surface_jacobian2Dw,
                                       nface_size*sizeof(float)),4305);
    // transfers data onto GPU
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_x_noise, normal_x_noise,
                                       nface_size*sizeof(float),hipMemcpyHostToDevice),4306);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_y_noise, normal_y_noise,
                                       nface_size*sizeof(float),hipMemcpyHostToDevice),4307);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_z_noise, normal_z_noise,
                                       nface_size*sizeof(float),hipMemcpyHostToDevice),4308);
    print_CUDA_error_if_any(hipMemcpy(mp->d_mask_noise, mask_noise,
                                       nface_size*sizeof(float),hipMemcpyHostToDevice),4309);
    print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_jacobian2Dw, free_surface_jacobian2Dw,
                                       nface_size*sizeof(float),hipMemcpyHostToDevice),4310);
  }

  // prepares noise strength kernel
  if( *NOISE_TOMOGRAPHY == 3 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_Sigma_kl),
                                       125*(mp->NSPEC_AB)*sizeof(float)),4401);
    // initializes kernel values to zero
    print_CUDA_error_if_any(hipMemset(mp->d_Sigma_kl,0,
                                       125*mp->NSPEC_AB*sizeof(float)),4403);

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //printf("jacobian_size = %d\n",25*(*num_free_surface_faces));
  exit_on_cuda_error("prepare_fields_noise_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// cleanup

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_cleanup_device,
              PREPARE_CLEANUP_DEVICE)(long* Mesh_pointer_f,
                                      int* SIMULATION_TYPE,
                                      int* SAVE_FORWARD,
                                      int* ACOUSTIC_SIMULATION,
                                      int* ELASTIC_SIMULATION,
                                      int* ABSORBING_CONDITIONS,
                                      int* NOISE_TOMOGRAPHY,
                                      int* COMPUTE_AND_STORE_STRAIN,
                                      int* ATTENUATION,
                                      int* OCEANS,
                                      int* APPROXIMATE_HESS_KL) {

TRACE("prepare_cleanup_device");

  // frees allocated memory arrays
  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // frees memory on GPU
  // mesh
  hipFree(mp->d_xix);
  hipFree(mp->d_xiy);
  hipFree(mp->d_xiz);
  hipFree(mp->d_etax);
  hipFree(mp->d_etay);
  hipFree(mp->d_etaz);
  hipFree(mp->d_gammax);
  hipFree(mp->d_gammay);
  hipFree(mp->d_gammaz);
  hipFree(mp->d_muv);

  // absorbing boundaries
  if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0 ){
    hipFree(mp->d_abs_boundary_ispec);
    hipFree(mp->d_abs_boundary_ijk);
    hipFree(mp->d_abs_boundary_normal);
    hipFree(mp->d_abs_boundary_jacobian2Dw);
  }

  // interfaces
  hipFree(mp->d_nibool_interfaces_ext_mesh);
  hipFree(mp->d_ibool_interfaces_ext_mesh);

  // global indexing
  hipFree(mp->d_ispec_is_inner);
  hipFree(mp->d_ibool);

  // sources
  if (*SIMULATION_TYPE == 1  || *SIMULATION_TYPE == 3){
    hipFree(mp->d_sourcearrays);
    hipFree(mp->d_stf_pre_compute);
  }

  hipFree(mp->d_islice_selected_source);
  hipFree(mp->d_ispec_selected_source);

  // receivers
  if( mp->nrec_local > 0 ) hipFree(mp->d_number_receiver_global);
  hipFree(mp->d_ispec_selected_rec);

  // ACOUSTIC arrays
  if( *ACOUSTIC_SIMULATION ){
    hipFree(mp->d_potential_acoustic);
    hipFree(mp->d_potential_dot_acoustic);
    hipFree(mp->d_potential_dot_dot_acoustic);
    hipFree(mp->d_send_potential_dot_dot_buffer);
    hipFree(mp->d_rmass_acoustic);
    hipFree(mp->d_rhostore);
    hipFree(mp->d_kappastore);
    hipFree(mp->d_phase_ispec_inner_acoustic);
    hipFree(mp->d_ispec_is_acoustic);

    if( *NOISE_TOMOGRAPHY == 0 ){
      hipFree(mp->d_free_surface_ispec);
      hipFree(mp->d_free_surface_ijk);
    }

    if( *ABSORBING_CONDITIONS ) hipFree(mp->d_b_absorb_potential);

    if( *SIMULATION_TYPE == 3 ) {
      hipFree(mp->d_b_potential_acoustic);
      hipFree(mp->d_b_potential_dot_acoustic);
      hipFree(mp->d_b_potential_dot_dot_acoustic);
      hipFree(mp->d_rho_ac_kl);
      hipFree(mp->d_kappa_ac_kl);
      if( *APPROXIMATE_HESS_KL) hipFree(mp->d_hess_ac_kl);
    }


    if(mp->nrec_local > 0 ){
      hipFree(mp->d_station_seismo_potential);
      free(mp->h_station_seismo_potential);
    }

  } // ACOUSTIC_SIMULATION

  // ELASTIC arrays
  if( *ELASTIC_SIMULATION ){
    hipFree(mp->d_displ);
    hipFree(mp->d_veloc);
    hipFree(mp->d_accel);
    hipFree(mp->d_send_accel_buffer);
    hipFree(mp->d_rmass);

    hipFree(mp->d_phase_ispec_inner_elastic);
    hipFree(mp->d_ispec_is_elastic);

    if( mp->nrec_local > 0 ){
      hipFree(mp->d_station_seismo_field);
      free(mp->h_station_seismo_field);
    }

    if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0){
      hipFree(mp->d_rho_vp);
      hipFree(mp->d_rho_vs);

      if(*SIMULATION_TYPE == 3 || ( *SIMULATION_TYPE == 1 && *SAVE_FORWARD ))
          hipFree(mp->d_b_absorb_field);
    }

    if( *SIMULATION_TYPE == 3 ) {
      hipFree(mp->d_b_displ);
      hipFree(mp->d_b_veloc);
      hipFree(mp->d_b_accel);
      hipFree(mp->d_rho_kl);
      hipFree(mp->d_mu_kl);
      hipFree(mp->d_kappa_kl);
      if( *APPROXIMATE_HESS_KL ) hipFree(mp->d_hess_el_kl);
    }

    if( *COMPUTE_AND_STORE_STRAIN ){
      hipFree(mp->d_epsilondev_xx);
      hipFree(mp->d_epsilondev_yy);
      hipFree(mp->d_epsilondev_xy);
      hipFree(mp->d_epsilondev_xz);
      hipFree(mp->d_epsilondev_yz);
      if( *SIMULATION_TYPE == 3 ){
        hipFree(mp->d_epsilon_trace_over_3);
        hipFree(mp->d_b_epsilon_trace_over_3);
        hipFree(mp->d_b_epsilondev_xx);
        hipFree(mp->d_b_epsilondev_yy);
        hipFree(mp->d_b_epsilondev_xy);
        hipFree(mp->d_b_epsilondev_xz);
        hipFree(mp->d_b_epsilondev_yz);
      }
    }

    if( *ATTENUATION ){
      hipFree(mp->d_factor_common);
      hipFree(mp->d_one_minus_sum_beta);
      hipFree(mp->d_alphaval);
      hipFree(mp->d_betaval);
      hipFree(mp->d_gammaval);
      hipFree(mp->d_R_xx);
      hipFree(mp->d_R_yy);
      hipFree(mp->d_R_xy);
      hipFree(mp->d_R_xz);
      hipFree(mp->d_R_yz);
      if( *SIMULATION_TYPE == 3){
        hipFree(mp->d_b_R_xx);
        hipFree(mp->d_b_R_yy);
        hipFree(mp->d_b_R_xy);
        hipFree(mp->d_b_R_xz);
        hipFree(mp->d_b_R_yz);
        hipFree(mp->d_b_alphaval);
        hipFree(mp->d_b_betaval);
        hipFree(mp->d_b_gammaval);
      }
    }

    if( *OCEANS ){
      if( mp->num_free_surface_faces > 0 ){
        hipFree(mp->d_rmass_ocean_load);
        hipFree(mp->d_free_surface_normal);
        hipFree(mp->d_updated_dof_ocean_load);
        if( *NOISE_TOMOGRAPHY == 0){
          hipFree(mp->d_free_surface_ispec);
          hipFree(mp->d_free_surface_ijk);
        }
      }
    }
  } // ELASTIC_SIMULATION

  // purely adjoint & kernel array
  if( *SIMULATION_TYPE == 2 || *SIMULATION_TYPE == 3 ){
    hipFree(mp->d_islice_selected_rec);
    if(mp->nadj_rec_local > 0 ){
      hipFree(mp->d_adj_sourcearrays);
      hipFree(mp->d_pre_computed_irec);
      free(mp->h_adj_sourcearrays_slice);
    }
  }

  // NOISE arrays
  if( *NOISE_TOMOGRAPHY > 0 ){
    hipFree(mp->d_free_surface_ispec);
    hipFree(mp->d_free_surface_ijk);
    hipFree(mp->d_noise_surface_movie);
    if( *NOISE_TOMOGRAPHY == 1 ) hipFree(mp->d_noise_sourcearray);
    if( *NOISE_TOMOGRAPHY > 1 ){
      hipFree(mp->d_normal_x_noise);
      hipFree(mp->d_normal_y_noise);
      hipFree(mp->d_normal_z_noise);
      hipFree(mp->d_mask_noise);
      hipFree(mp->d_free_surface_jacobian2Dw);
    }
    if( *NOISE_TOMOGRAPHY == 3 ) hipFree(mp->d_Sigma_kl);
  }

  // mesh pointer - not needed anymore
  free(mp);
}

