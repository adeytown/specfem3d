#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <mpi.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_stacey_elastic_kernel(real* veloc, 
                                              real* accel, 
                                              real* b_accel, 
                                              int* abs_boundary_ispec,
                                              int* abs_boundary_ijk, int* ibool,
                                              real* abs_boundary_normal,
                                              real* rho_vp, real* rho_vs,
                                              real* abs_boundary_jacobian2Dw,
                                              real* b_absorb_field,
                                              int* ispec_is_inner, int* ispec_is_elastic,
                                              int phase_is_inner,float* debug_val,int* debug_val_int,
                                              int num_abs_boundary_faces,
                                              int SAVE_FORWARD,int SIMULATION_TYPE) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx
  int i;
  int j;
  int k;
  int iglob;
  int ispec;
  realw vx,vy,vz,vn;
  realw nx,ny,nz;
  realw rho_vp_temp,rho_vs_temp;
  realw tx,ty,tz;
  realw jacobianw;
  
  // don't compute points outside NGLLSQUARE == NGLL2 == 25
  if(igll < NGLL2 && iface < num_abs_boundary_faces) {    
    
    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;
    i = abs_boundary_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)]-1;
    j = abs_boundary_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)]-1;
    k = abs_boundary_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)]-1;
    iglob = ibool[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;
    
    if(ispec_is_inner[ispec] == phase_is_inner && ispec_is_elastic[ispec]==1) {

      i = abs_boundary_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)]-1;
      j = abs_boundary_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)]-1;
      k = abs_boundary_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)]-1;
      iglob = ibool[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;
      
      // gets associated velocity
      
      vx = veloc[iglob*3+0];
      vy = veloc[iglob*3+1];
      vz = veloc[iglob*3+2];
      
      // gets associated normal
      nx = abs_boundary_normal[INDEX3(NDIM,NGLL2,0,igll,iface)];
      ny = abs_boundary_normal[INDEX3(NDIM,NGLL2,1,igll,iface)];
      nz = abs_boundary_normal[INDEX3(NDIM,NGLL2,2,igll,iface)];
      
      // // velocity component in normal direction (normal points out of element)
      vn = vx*nx + vy*ny + vz*nz;
      rho_vp_temp = rho_vp[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];
      rho_vs_temp = rho_vs[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];
      tx = rho_vp_temp*vn*nx + rho_vs_temp*(vx-vn*nx);
      ty = rho_vp_temp*vn*ny + rho_vs_temp*(vy-vn*ny);
      tz = rho_vp_temp*vn*nz + rho_vs_temp*(vz-vn*nz);
      
      jacobianw = abs_boundary_jacobian2Dw[INDEX2(NGLL2,igll,iface)];            
   
      atomicAdd(&accel[iglob*3],-tx*jacobianw);
      atomicAdd(&accel[iglob*3+1],-ty*jacobianw);
      atomicAdd(&accel[iglob*3+2],-tz*jacobianw);

      if(SIMULATION_TYPE == 3) {
	atomicAdd(&b_accel[iglob*3  ],-b_absorb_field[0+3*(igll+25*(iface))]);
	atomicAdd(&b_accel[iglob*3+1],-b_absorb_field[1+3*(igll+25*(iface))]);
	atomicAdd(&b_accel[iglob*3+2],-b_absorb_field[2+3*(igll+25*(iface))]);
      }
      else if(SAVE_FORWARD && SIMULATION_TYPE == 1) {
	b_absorb_field[0+3*(igll+25*(iface))] = tx*jacobianw;
	b_absorb_field[1+3*(igll+25*(iface))] = ty*jacobianw;
	b_absorb_field[2+3*(igll+25*(iface))] = tz*jacobianw;
      }
      
    }
  }

}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(compute_stacey_elastic_cuda,
              COMPUTE_STACEY_ELASTIC_CUDA)(long* Mesh_pointer_f, 
                                           int* NSPEC_ABf, 
                                           int* NGLOB_ABf, 
                                           int* phase_is_innerf, 
                                           int* num_abs_boundary_facesf, 
                                           int* SIMULATION_TYPEf, 
                                           int* NSTEPf, 
                                           int* NGLOB_ADJOINTf, 
                                           int* b_num_abs_boundary_facesf, 
                                           int* b_reclen_fieldf,
                                           float* b_absorb_field, 
                                           int* SAVE_FORWARDf, 
                                           int* itf) {

TRACE("compute_stacey_elastic_cuda");  
  
  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container
  //int fid = 0;
  //int it = *itf;
  //int NSPEC_AB = *NSPEC_ABf;
  //int NGLOB_AB = *NGLOB_ABf;
  int phase_is_inner	     = *phase_is_innerf;
  int num_abs_boundary_faces     = *num_abs_boundary_facesf;
  int SIMULATION_TYPE	     = *SIMULATION_TYPEf;
  //int NSTEP			     = *NSTEPf;
  int myrank; MPI_Comm_rank(MPI_COMM_WORLD,&myrank);
  //int NGLOB_ADJOINT		     = *NGLOB_ADJOINTf;
  //int b_num_abs_boundary_faces   = *b_num_abs_boundary_facesf;
  int b_reclen_field	     = *b_reclen_fieldf;
  int SAVE_FORWARD             = *SAVE_FORWARDf;              

  int blocksize = 32; // > NGLL2=25, but we handle this inside kernel
  int num_blocks_x = num_abs_boundary_faces;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = ceil(num_blocks_x/2.0);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  float* d_debug_val;
  int* d_debug_val_int;

  if(SIMULATION_TYPE == 3 && num_abs_boundary_faces > 0) {
    // int val = NSTEP-it+1;
    // read_abs_(&fid,(char*)b_absorb_field,&b_reclen_field,&val);    
    // The read is done in fortran
    hipMemcpy(mp->d_b_absorb_field,b_absorb_field,b_reclen_field,hipMemcpyHostToDevice);
  }
  
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("between cudamemcpy and compute_stacey_elastic_kernel");
#endif
  
  compute_stacey_elastic_kernel<<<grid,threads>>>(mp->d_veloc,mp->d_accel,mp->d_b_accel,
                                                  mp->d_abs_boundary_ispec, mp->d_abs_boundary_ijk, 
                                                  mp->d_ibool, 
                                                  mp->d_abs_boundary_normal, 
                                                  mp->d_rho_vp, mp->d_rho_vs, 
                                                  mp->d_abs_boundary_jacobian2Dw, 
                                                  mp->d_b_absorb_field,
                                                  mp->d_ispec_is_inner, 
                                                  mp->d_ispec_is_elastic, 
                                                  phase_is_inner,
                                                  d_debug_val,d_debug_val_int,
                                                  num_abs_boundary_faces,
                                                  SAVE_FORWARD,SIMULATION_TYPE);
  
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_stacey_elastic_kernel");  
#endif

  // ! adjoint simulations: stores absorbed wavefield part
  // if (SIMULATION_TYPE == 1 .and. SAVE_FORWARD .and. num_abs_boundary_faces > 0 ) &
  //   write(IOABS,rec=it) b_reclen_field,b_absorb_field,b_reclen_field
  
  if(SIMULATION_TYPE==1 && SAVE_FORWARD && num_abs_boundary_faces>0) {
    hipMemcpy(b_absorb_field,mp->d_b_absorb_field,b_reclen_field,hipMemcpyDeviceToHost);
    // The write is done in fortran
    // write_abs_(&fid,(char*)b_absorb_field,&b_reclen_field,&it);    
  }
  
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after compute_stacey_elastic after cudamemcpy");  
#endif
}

